/* 
    Parallel Programming Matrix Multiplication.
    Parallelized version of matrix multiplication in CUDA and MPI.
    CUDA Code
*/

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

#define BLOCK_SIZE 16

__global__ void matrix_mult_kernel(float* matrixOne, float* matrixTwo, float* resultMatrix, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < m && col < k) {
        float sum = 0.0f;
        for (int i = 0; i < n; i++) {
            // Calculate the running sum for the current result matrix position
            sum += matrixOne[row * n + i] * matrixTwo[i * k + col];
        }
        resultMatrix[row * k + col] = sum;
    }
}

extern "C" {
    void matrix_mult_cuda(float* matrixOne, float* matrixTwo, float* resultMatrix, int m, int n, int k) {
        // Alocate memory for 3 matrices
        float* d_matrixOne, * d_matrixTwo, * d_resultMatrix;
        hipMallocManaged(&d_matrixOne, m * n * sizeof(float));
        hipMallocManaged(&d_matrixTwo, n * k * sizeof(float));
        hipMallocManaged(&d_resultMatrix, m * k * sizeof(float));

        // Copy the values of the 2 given matricies from device to host
        hipMemcpy(d_matrixOne, matrixOne, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_matrixTwo, matrixTwo, n * k * sizeof(float), hipMemcpyHostToDevice);

        // Setting up the grid size and call the CUDA function to do matrix multiplications
        int blockSize = BLOCK_SIZE;
        // See https://docs.nvidia.com/cuda/cuda-c-programming-guide/#dim3. Integer vector type to specify dimensions. Initializes unspecified components to 1.
        dim3 gridSize((k + blockSize - 1) / blockSize, (m + blockSize - 1) / blockSize);
        matrix_mult_kernel<<<gridSize, dim3(blockSize, blockSize)>>>(d_matrixOne, d_matrixTwo, d_resultMatrix, m, n, k);

        // Wait until all GPU work is finished
        hipDeviceSynchronize();

        // Copies results of the matrix multiplication from device (d_resultMatrix) to host (resultMatrix)
        hipMemcpy(resultMatrix, d_resultMatrix, m * k * sizeof(float), hipMemcpyDeviceToHost);

        // Free all 3 matrices
        hipFree(d_matrixOne);
        hipFree(d_matrixTwo);
        hipFree(d_resultMatrix);
    }

    // Fill matrix with random values using cuRAND
    void matrix_init(float* matrix, int size, unsigned int seed, int offset) {
        hiprandGenerator_t gen;
        // Create random generator instance
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        // Set random generator set
        hiprandSetPseudoRandomGeneratorSeed(gen, seed + offset);
        // Make sure generated values follow uniform/normal ditribution
        hiprandGenerateUniform(gen, matrix, size);
        // End random generator instance
        hiprandDestroyGenerator(gen);
    }

    // Display matrix in table with rows and columns.
    void print_matrix(float* matrix, int rows, int cols) {
        for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {
                printf("%.2f ", matrix[i * cols + j]);
            }
            printf("\n");
        }
        printf("\n");
    }
}